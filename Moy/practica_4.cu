#include "hip/hip_runtime.h"
/********************************************************
 *														*
 * Autor: Programacion_en_CUDA-Moises Arreola Zamora	*
 *														*
 * Fecha:12/10/2020										*
 * 														*
 * Lanzamiento de Kernel								*
 *														*
 * 														*
 * compilacion con: 									*
 * nvcc fractal.cu -o fractal.out -lglut -lGL -lGLU		*
*********************************************************/
//librerias

#include <stdio.h>
#include <stdlib.h>
#include ""
#include <hip/hip_runtime.h>

//definiciones
#define NUM_MATRICES	3
#define PROFUNDIDAD		24

#define PRIMER_SEG		24
#define	SEGUNDO_SEG		48
#define TERCER_SEG		72


__global__ void rellenado( int *array) {
	int id_global 	= threadIdx.x + blockDim.x * blockIdx.x;
	int id_thread 	= threadIdx.x;
	int id_block 	= blockIdx.x;
	//no usar for, apoyarse del no de bloques o el no de threads	
	if (id_global < PRIMER_SEG){
		array[id_global] = id_thread;
	}
	else if(id_global < SEGUNDO_SEG){
		array[id_global] = id_block;
	}
	else
		array[id_global] = id_global;
	
}

// MAIN:
int main(int argc, char** argv)
{
// declaracion var host
int i;
int bloques = 0;
int hilos 	= 0;
int *host_x;

//var device
int *dev_array;

//reserva al host
host_x = (int*)malloc(NUM_MATRICES*PROFUNDIDAD*sizeof(int));

// reserva en el device
hipMalloc( (void**)&dev_array, NUM_MATRICES*PROFUNDIDAD*sizeof(int) );

//ingreso de valores 
printf("Ejecutando Practica 4\n");

//invocacion kernel
for (i = 0; i < NUM_MATRICES; i++){
	if(i == 0){
		bloques	=	1;
		hilos	=	24;
		//~ rellenado<<1,24>>(dev_array);
	}
	else if( i == 1){
		bloques	=	24;
		hilos	=	1;
		//~ rellenado<<bloques,hilos>>(dev_array);
	}
	else {
		bloques	=	4;
		hilos	=	6;
		//~ rellenado<<bloques,hilos>>(dev_array);
	}
	printf("bloques %d hilos %d", bloques,hilos);
	//~ rellenado<<bloques,hilos>>(dev_array);
}
rellenado<<NUM_MATRICES,PRIMER_SEG>> (dev_array);
//~ add<<<N, 1>>> (dev_a, dev_b, dev_c);
//copia de device a host
hipMemcpy(host_x, dev_array, NUM_MATRICES*PROFUNDIDAD*sizeof(float),hipMemcpyDeviceToHost);

// salida

	
hipFree(dev_array);
//~ free(host_x);
printf("\nFin de Programa \n");

fflush(stdin);
return 0;
}

