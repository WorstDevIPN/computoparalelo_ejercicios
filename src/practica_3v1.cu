/********************************************************
 *														*
 * Autor: Programacion_en_CUDA-Moises Arreola Zamora	*
 *														*
 * Fecha:12/10/2020										*
 * 														*
 * Lanzamiento de Kernel								*
 *														*
 * 														*
 * compilacion con: 									*
 * nvcc fractal.cu -o fractal.out -lglut -lGL -lGLU		*
*********************************************************/

// includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void chicharron(int a, int b, int c, float *dev_x) {
	//distincion de raiz imaginaria
	int ima_x = (b*b)-(4*a*c);
	
	if (ima_x < 0){
		ima_x = -1 * ima_x;
		dev_x[0] = ((sqrtf(ima_x))/(2*a));
		dev_x[1] = (-1)*((sqrtf(ima_x))/(2*a));
		return;
	}
	else{
		//~ real_p = (float)0;
		dev_x[0] = ((-b)/(2*a))+((sqrtf(ima_x))/(2*a));
		dev_x[1] = ((-b)/(2*a))-((sqrtf(ima_x))/(2*a));
		return;
	}

}


// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{
// declaracion var host
int coef_a;
int coef_b;
int coef_c;
float host_real;

float *host_x;

//var device
float *dev_x;

//reserva al host
host_x = (float*)malloc(2*sizeof(float));

// reserva en el device
hipMalloc( (void**)&dev_x, 2*sizeof(float) );

//ingreso de valores 
printf("Ingresa los coeficientes de la ecuacion ax^2 + bx + c = 0\n");
printf("Coeficiente a\t");
scanf("%d",&coef_a);
printf("Coeficiente b\t");
scanf("%d",&coef_b);        
printf("Coeficiente c\t");
scanf("%d",&coef_c);
printf("\n");
host_real =(-(float)coef_b)/(2*(float)coef_a);

//invocacion kernel		
chicharron<<<1, 1>>> (coef_a, coef_b, coef_c, dev_x);	

//copia de device a host
hipMemcpy(host_x, dev_x, 2*sizeof(float),hipMemcpyDeviceToHost);

// salida
if((coef_b*coef_b)-(4*coef_a*coef_c) < 0){
	
	printf("Las soluciones son:\n x1 = %f %f i\t x2 = %f %f i", host_real, host_x[0], host_real, host_x[1]);
}
else
	printf("Las soluciones son:\n x1 = %f \t x2 = %f ", host_x[0], host_x[1]);
	
hipFree( dev_x );
free(host_x);
printf("\nFin de Programa \n");

fflush(stdin);
return 0;
}
