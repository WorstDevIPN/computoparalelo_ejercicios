#include "hip/hip_runtime.h"
/************************************************************\
 *															*
 * Titulo			:	Creación de eventos					*
 * Organizacion		:	CIC-IPN								*
 * Autor	       	:	Oswaldo Franco						*
 * e-mail			:	oswaldo1509@gmail.com				*
 * Periodo			:	Semestre B20						*
 * Dependencies		:	hipError_t.h							*
 *															*
\************************************************************/

#include<stdio.h>
#include<stdlib.h>
#include"hipError_t.h"

#define nHilos	4

__global__ void reverse(int *org, int *rev, int N){
	int global = threadIdx.x + blockDim.x * blockIdx.x;
	rev[(N-1)-global] = org[global];
}

int main(){
	int N=0;
	int *hst_arr, *hst_rev;
	int *dev_arr, *dev_rev;
	float elapsedTime;
	hipEvent_t start, stop;
	
	printf("Ingresa N ");
	scanf("%d", &N);

	hst_arr = (int*)malloc( N*sizeof(float));
	hst_rev = (int*)malloc( N*sizeof(float));

	hipMalloc( (void**)&dev_arr, N*sizeof(int) );
	hipMalloc( (void**)&dev_rev, N*sizeof(int) );

	srand(time(NULL));

	for(int i=0; i<N; i++){
		hst_arr[i] = rand() % 9;
	}
	
	hipMemcpy(dev_arr, hst_arr, N*sizeof(int),hipMemcpyHostToDevice);
	check_CUDA_Error("ERROR EN hipMemcpy");

	int nBloques = N / nHilos;

	nBloques = (N%nHilos==0) ? nBloques : nBloques+1;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);
	reverse<<< nBloques, nHilos >>>(dev_arr, dev_rev, N);
	check_CUDA_Error("ERROR EN reverse");
	hipEventRecord(stop,0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Tiempo de ejecucion en GPU %f ms\n",elapsedTime);

	hipMemcpy(hst_rev, dev_rev, N*sizeof(int),hipMemcpyDeviceToHost);
	check_CUDA_Error("ERROR EN hipMemcpy");

	printf("arreglo original\n");
	for(int i=0; i<N; i++){
		printf("%d\t",hst_arr[i]);
	}
	printf("\n");

	printf("arreglo en reversa\n");
	for(int i=0; i<N; i++){
		printf("%d\t",hst_rev[i]);
	}
	printf("\n");

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(dev_arr);
	hipFree(dev_rev);
	free(hst_arr);
	free(hst_rev);
	return 0;
}
