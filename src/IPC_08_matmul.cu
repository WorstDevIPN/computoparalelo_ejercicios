#include "hip/hip_runtime.h"
/****************************************************************\
 *																*
 * Titulo			:	Uso de memoria constante				*
 * Organizacion		:	CIC-IPN									*
 * Autor	       	:	Oswaldo Franco							*
 * e-mail			:	oswaldo1509@gmail.com					*
 * Periodo			:	Semestre B20							*
 * Dependencies		:	hipError_t.h								*
 *																*
\****************************************************************/

#include<stdio.h>
#include<stdlib.h>

#define N 4

__constant__ float dev_A[N][N];
__constant__ float dev_B[N][N];


__global__ void matmul(float *C){
	int columna = threadIdx.x;
	int fila = threadIdx.y;
	int pos = columna + fila * N;
	
	for (int i=0; i<N; i++){
		C[pos] += dev_A[fila][i] * dev_B[i][columna];
	}
}

int main(){
	float *hst_A, *hst_B, *hst_C;
	float *dev_C;
	dim3 Nbloques(1);
	dim3 hilosB(N,N);
	hipDeviceProp_t myGPU;
	hipEvent_t start, stop;
	float elapsedTime;
	size_t memLibre, memTotal;
	

	srand(time(NULL));

	hst_A = (float*)malloc(N*N*sizeof(float));
	hst_B = (float*)malloc(N*N*sizeof(float));
	hst_C = (float*)malloc(N*N*sizeof(float));

	hipMalloc ((void**)&dev_C, N*N*sizeof(float));

	for(int i=0; i<N*N; i++){
		hst_A[i] = (float) (rand()%2);
		hst_B[i] = (float) (rand()%2);
	}

	hipGetDeviceProperties(&myGPU, 0);
	printf("totalConstMem: %zu\n", myGPU.totalConstMem);

	hipMemcpyToSymbol(HIP_SYMBOL(dev_A), hst_A, N*N*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_B), hst_B, N*N*sizeof(float));

	hipMemGetInfo(&memLibre, &memTotal);
	printf("memLibre: %zu\n", memLibre);


	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);

	matmul<<< Nbloques, hilosB >>>(dev_C);

	hipEventRecord(stop,0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Elapsed Time: %.2f\n",elapsedTime);

	hipMemcpy(hst_C, dev_C, N*N*sizeof(float), hipMemcpyDeviceToHost);

	for(int i=0; i<N; i++){
		for(int j=0; j<N; j++){
			printf("%.2f\t",hst_A[j + i*N]);
		}
		printf("\n");
	}
	printf("\n");

	for(int i=0; i<N; i++){
		for(int j=0; j<N; j++){
			printf("%.2f\t",hst_B[j + i*N]);
		}
		printf("\n");
	}
	printf("\n");

	for(int i=0; i<N; i++){
		for(int j=0; j<N; j++){
			printf("%.2f\t",hst_C[j + i*N]);
		}
		printf("\n");
	}
	printf("\n");

	hipFree(dev_C);
	return 0;
}
