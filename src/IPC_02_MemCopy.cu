/************************************************************\
 *															*
 * Titulo			:	Memoria de la GPU					*
 * Organizacion		:	CIC-IPN								*
 * Autor	       	:	Oswaldo Franco						*
 * e-mail			:	oswaldo1509@gmail.com				*
 * Periodo			:	Semestre B20						*
 * Dependencies		:										*
 *															*
\************************************************************/


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>

#define N 8

int main(){
	float *hst_matrixA;
	float *hst_matrixB;

	float *dev_matrixA;
	float *dev_matrixB;
	
	size_t memLibre, memTotal;

	hst_matrixA = (float *)malloc( N*N*sizeof(float) );
	hst_matrixB = (float *)malloc( N*N*sizeof(float) );
	hipMalloc ( (void**)&dev_matrixA, N*N*sizeof(float) );
	hipMalloc ( (void**)&dev_matrixB, N*N*sizeof(float) );

	srand( (int)time(NULL) );

	for(int i=0; i<N*N; i++){
		hst_matrixA[i] = (float)(rand()%10);
	}

	hipMemcpy(dev_matrixA, hst_matrixA, N*N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_matrixB, dev_matrixA, N*N*sizeof(float), hipMemcpyDeviceToDevice);
	hipMemcpy(hst_matrixB, dev_matrixB, N*N*sizeof(float), hipMemcpyDeviceToHost);

	printf("======== Matriz Hst A ===========\n");

	for(int i=0; i<N*N; i++){
		if (i%8 == 0) printf("\n");
		printf("%.2f \t ", hst_matrixA[i]);
	}
	printf("\n");

	printf("======== Matriz Hst B ===========\n");

	for(int i=0; i<N*N; i++){
		if (i%8 == 0) printf("\n");
		printf("%.2f \t ", hst_matrixB[i]);
	}

	printf("\n");

	hipMemGetInfo( &memLibre, &memTotal);
	printf("Libre %zu MB Total %zu MB \n", memLibre/1024, memTotal/1024);

	hipFree(dev_matrixA);
	hipFree(dev_matrixB);

	return 0;
}
