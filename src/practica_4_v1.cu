/********************************************************
 *														*
 * Autor: Programacion_en_CUDA-Moises Arreola Zamora	*
 *														*
 * Fecha:12/10/2020										*
 * 														*
 * Lanzamiento de Kernel con multiples hilos y bloques	*
 *														*
 * 														*
 * compilacion con: 									*
 * nvcc practica_4.cu -o practica_4.o				 	*
*********************************************************/
//librerias

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

//definiciones
#define NUM_MATRICES	3
#define PROFUNDIDAD		24

#define PRIMER_SEG		24
#define	SEGUNDO_SEG		48
#define TERCER_SEG		72


__global__ void rellenado( int *array_threads, int *array_blocks, int *array_global) {
	int id_global 	= threadIdx.x + (blockDim.x * blockIdx.x);
	int id_thread 	= threadIdx.x;
	int id_block 	= blockIdx.x;
	
	//no usar for, apoyarse del no de bloques o el no de threads
	array_threads[id_global] = id_thread;
	array_blocks[id_global] = id_block;
	array_global[id_global] = id_global;

}

__host__ void desplegar_datos(int *thread, int *block, int *global){
	int j;
	
	for (j=0; j < NUM_MATRICES*PROFUNDIDAD; j++){
		if (j < PRIMER_SEG){
			printf("Arreglo_hilo [%d] \t%d\n", j, thread[j]);
		}
		else if(PRIMER_SEG <= j && j < SEGUNDO_SEG){
			if (j == PRIMER_SEG) printf("\n");
			printf("Arreglo_bloque [%d] \t%d\n", j, block[j-24]);
		}
		else if(j >= SEGUNDO_SEG && j < TERCER_SEG){
			if (j == SEGUNDO_SEG) printf("\n");
			printf("Array_global [%d] \t%d\n", j, global[j-48]);
		}
	}
}

// MAIN:
int main(int argc, char** argv)
{
	// declaracion var host
	int i;
	int bloques = 0;
	int hilos 	= 0;
	int *host_thread;
	int *host_block;
	int *host_global;
	//~ cudaDeviceProp myGPU;
	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	//var device
	int *dev_array_global;
	int *dev_array_threads;
	int *dev_array_blocks;


	//reserva al host
	host_thread = (int*)malloc(PROFUNDIDAD*sizeof(int));
	host_block = (int*)malloc(PROFUNDIDAD*sizeof(int));
	host_global= (int*)malloc(PROFUNDIDAD*sizeof(int));

	// reserva en el device
	hipMalloc( (void**)&dev_array_global, PROFUNDIDAD*sizeof(int) );
	hipMalloc( (void**)&dev_array_threads, PROFUNDIDAD*sizeof(int) );
	hipMalloc( (void**)&dev_array_blocks, PROFUNDIDAD*sizeof(int) );

	//ingreso de valores 
	printf("Ejecutando Practica 4\n");

	//invocacion kernel
	hipEventRecord(start,0);
	for (i = 0; i < NUM_MATRICES; i++){
		if(i == 0){
			bloques	=	1;
			hilos	=	24;
			//~ rellenado<<1,24>>(dev_array);
		}
		else if( i == 1){
			bloques	=	24;
			hilos	=	1;
			//~ rellenado<<bloques,hilos>>(dev_array);
		}
		else {
			bloques	=	4;
			hilos	=	6;
			//~ rellenado<<bloques,hilos>>(dev_array);
		}
		printf("\nbloques %d hilos %d\n\n", bloques,hilos);
		
		rellenado<<<bloques,hilos>>>(dev_array_threads, dev_array_blocks, dev_array_global);
		
		//~ //copia de device a host
		//~ cudaMemcpy(host_x, dev_array_threads, NUM_MATRICES*PROFUNDIDAD*sizeof(int),cudaMemcpyDeviceToHost);
		hipDeviceSynchronize();
		//copia de device a host
		hipMemcpy(host_thread, dev_array_threads, PROFUNDIDAD*sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(host_block, dev_array_blocks, PROFUNDIDAD*sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(host_global, dev_array_global, PROFUNDIDAD*sizeof(int),hipMemcpyDeviceToHost);
		//~ // salida	
		desplegar_datos(host_thread, host_block, host_global);
	}
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Elapsed Time: %.3f\n",elapsedTime);

	//memory dump
	hipFree(dev_array_threads);
	hipFree(dev_array_blocks);
	hipFree(dev_array_global);
	
	free(host_thread);
	free(host_block);
	free(host_global);
	
	printf("\nFin de Programa \n");

	fflush(stdin);
	return 0;
}

