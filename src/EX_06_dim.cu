#include "hip/hip_runtime.h"
/****************************************************************\
 *																*
 * Titulo			:	Ejecucion de hilos multidimensionales	*
 * Organizacion		:	CIC-IPN									*
 * Autor	       	:	Oswaldo Franco							*
 * e-mail			:	oswaldo1509@gmail.com					*
 * Periodo			:	Semestre B20							*
 * Dependencies		:	hipError_t.h								*
 *																*
\****************************************************************/

#include<stdio.h>
#include<stdio.h>
#include"hipError_t.h"

#define	N 16

__device__ int index(int columna, int fila){
	return columna + fila * N;
}

__global__ void kernel(int *org, int *mod){

	int columna = threadIdx.x;
	int fila = threadIdx.y;
	int id = columna + fila * blockDim.x;

	//printf("global %d \t", global);

	if (columna%N>0 && columna%N<N-1 && fila%N>0 && fila<N-1){

		mod[id] = org[index(columna-1, fila)] + org[index(columna+1, fila)] + org[index(columna,fila-1)] + org[index(columna, fila+1)];
	}
	else{
		mod[id] = org[id];
	}
}

int main(){
	int *hst_matrixA, *hst_matrixB;
	int *dev_matrixA, *dev_matrixB;
	dim3 Nbloques(1);
	dim3 hilosB(N,N);
	hipEvent_t start, stop;
	float elapsedTime;
	
	srand(time(NULL));

	hst_matrixA = (int*)malloc(N*N*sizeof(int));
	hst_matrixB = (int*)malloc(N*N*sizeof(int));
	hipMalloc((void**)&dev_matrixA,N*N*sizeof(int));
	hipMalloc((void**)&dev_matrixB,N*N*sizeof(int));

	for(int i=0; i<N*N; i++)
		hst_matrixA[i] = rand() % 2;

	hipEventCreate(&start);
	hipEventCreate(&stop);


	hipMemcpy(dev_matrixA, hst_matrixA, N*N*sizeof(int), hipMemcpyHostToDevice);
	check_CUDA_Error("ERROR de Memcpy");

	hipEventRecord(start,0);

	kernel<<<Nbloques, hilosB >>>(dev_matrixA, dev_matrixB);
	check_CUDA_Error("ERROR de Kernel");

	hipEventRecord(stop,0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);


	hipMemcpy(hst_matrixB, dev_matrixB, N*N*sizeof(int), hipMemcpyDeviceToHost);
	check_CUDA_Error("ERROR de Memcpy");

	printf("Tiempo de ejecucion %.8f\n",elapsedTime);

	for(int i=0; i<N*N; i++){
		if (i%N == 0) printf("\n");
		printf("%d \t", hst_matrixA[i]);
	}
	printf("\n");

	for(int i=0; i<N*N; i++){
		if (i%N == 0) printf("\n");
		printf("%d \t", hst_matrixB[i]);
	}
	printf("\n");
	
	hipFree(dev_matrixA);
	hipFree(dev_matrixB);

	return 0;
}
