#include "hip/hip_runtime.h"
// bug en ordenamiento
// Data to merge: {0,0,1,4,4,3,1,3,},{2,2,0,0,6,0,1,1,}
// KernelOnGPU Time elapsed 0.000007 sec
// Merged Data: {0,0,0,0,1,1,1,1,0,2,2,3,3,4,4,6,}

#include "common.h"
#include<stdio.h>
#include<stdlib.h>

#include"book.h"
#define N 16

__device__ int mr[2*N];

__global__ void merge(int *left, int *right, int *merged, int size)	{

	int i = 0, j = 0, k = 0;

	while ( i<size/2 && j<size/2 ) {
		if ( left[i] <= right[j] ) {
			mr[k] = left[i];
			i ++;
		}
		else {
			mr[k] = right[j];
			j ++;
		}
		k++;
	}
	while ( i<size/2 ) {
		mr[k] = left[i];
		i ++;
		k ++;
	}
	while ( j<size/2 ){
		mr[k] = right[j];
		j ++;
		k ++;
	}

	for (int i=0; i<2*size; i++){
		merged[i] = mr[i];
	}

}


__global__ void mergesort (int *left, int *right,  int size)	{
	int tid = threadIdx.x;
	int dsize = size >> 1;

	if ( size == 1) {
		return;
	}
	else {	
        int *lleft;
        int *lright;
		hipMalloc((void**)&lleft, sizeof(int) * dsize);
		hipMalloc((void**)&lright, sizeof(int) * dsize);
		if (tid == 0){
            for(int i = 0; i < dsize; i++) {
                lleft[i] = left[i];
                lright[i] = left[dsize + i];
            }          
			mergesort<<<1,2>>>(lleft, lright, dsize);
			hipDeviceSynchronize();
			merge<<<1,1>>>(lleft, lright, left, size);
		}
		else if (tid == 1){
            for(int i = 0; i < dsize; i++) {
                lleft[i] = right[i];
                lright[i] = right[dsize + i];
            }          
			mergesort<<<1,2>>>(lleft, lright, dsize);
			hipDeviceSynchronize();
			merge<<<1,1>>>(lleft, lright, right, size);
		}
	}
}


int main (void)	{
	int *l;
	int *r;
	int *m;
	int *dev_l, *dev_r, *dev_m;
	int hi=2*N, lo=0;
    double iStart, iElaps;
    
    
    l = (int*)malloc(sizeof(int) * N);
    r = (int*)malloc(sizeof(int) * N);
    m = (int*)malloc(sizeof(int) * 2 * N);
    
	
	srand(time(NULL));

	HANDLE_ERROR( hipMalloc( (void**)&dev_l, N*sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_r, N*sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_m, 2*N*sizeof(int) ) );

    for(int i = 0; i < N; i++) {
        l[i] = rand() % (hi - lo) + 1;
        r[i] = rand() % (hi - lo) + 1;
    }

	printf("Data to merge: {");
	for (int i=0; i<N; i++)	{
		printf("%d,", l[i]);
	}
	printf("},{");
	for (int i=0; i<N; i++)	{
		printf("%d,", r[i]);
	}
	printf("}\n");


	HANDLE_ERROR( hipMemcpy( dev_l, l, N*sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy( dev_r, r, N*sizeof(int), hipMemcpyHostToDevice) );

    iStart = seconds();
	mergesort<<<1,2>>>(dev_l, dev_r, N);
	hipDeviceSynchronize();
	merge<<<1,1>>>(dev_l, dev_r, dev_m, 2*N);
    iElaps = seconds() - iStart;
	
	HANDLE_ERROR( hipMemcpy( m, dev_m, 2*N*sizeof(int),hipMemcpyDeviceToHost) );

	printf("Merged Data: {");
	for (int i=0; i<N; i++)	{
		printf("%d,%d,", m[2*i], m[2*i+1]);
	}
	printf("}\n");

    printf("KernelOnGPU Time elapsed %f sec\n",iElaps);
	
	free(l);
	free(r);
	free(m);

	hipFree( dev_l );
	hipFree( dev_r );
	hipFree( dev_m );
	hipDeviceSynchronize();
}

