// includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#define N 8

//~ __global__ void copy(float* dev_A_matriz, float* dev_B_matriz) {
   //~ cudaMemcpy(dev_B_matriz, dev_A_matriz, N*N*sizeof(float),  //operaciones de memoria 
															//no se permiten en devices
		//~ cudaMemcpyDeviceToDevice);
//~ }


// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{
// declaracion
float *hst_A_matriz;
float *hst_B_matriz;
float *dev_A_matriz;
float *dev_B_matriz;
size_t 	memlibre = 0;
size_t 	memtotal = 0;
// reserva en el host
hst_A_matriz = (float*)malloc( N*N*sizeof(float) );
hst_B_matriz = (float*)malloc( N*N*sizeof(float) );
// reserva en el device
hipMalloc( (void**)&dev_A_matriz, N*N*sizeof(float) );
hipMalloc( (void**)&dev_B_matriz, N*N*sizeof(float) );
// inicializacion de datos
srand ( (int)time(NULL) );
for (int i=0; i<N*N; i++){
	hst_A_matriz[i] = (float)( rand() % 2 );
}
// copia de datos
hipMemcpy(dev_A_matriz, hst_A_matriz, N*N*sizeof(float), 
		hipMemcpyHostToDevice);

// copia de datos entre dispositivos
//~ copy<<<1, 1>>> (dev_A_matriz, dev_B_matriz);
hipMemcpy(dev_B_matriz, dev_A_matriz, N*N*sizeof(float), 
		hipMemcpyDeviceToDevice);

//copia de device a host
hipMemcpy(hst_B_matriz, dev_B_matriz, N*N*sizeof(float), 
		hipMemcpyDeviceToHost);
// salida
for (int i=0; i<N*N; i++){
	if (hst_A_matriz[i]-hst_B_matriz[i]==0){
		printf("El valor Matriz A[%d] es igual a B[%d] %.2f - %.2f = %.2f \n",
			i,i,hst_A_matriz[i],hst_B_matriz[i],hst_A_matriz[i]-hst_B_matriz[i]);
	}
}
hipMemGetInfo(&memlibre,&memtotal);
printf("La memoria libre es de %zu y la memoria total es de %zu",memlibre,memtotal);
hipFree( dev_A_matriz );
hipFree( dev_B_matriz );
printf("\npulsa INTRO para finalizar...");
fflush(stdin);
char tecla = getchar();
return 0;
}
