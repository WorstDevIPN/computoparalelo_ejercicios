/************************************************************\
 *															*
 * Titulo			:	Propiedades de GPU					*
 * Organizacion		:	CIC-IPN								*
 * Autor	       	:	Oswaldo Franco						*
 * e-mail			:	oswaldo1509@gmail.com				*
 * Periodo			:	Semestre B20						*
 * Dependencies		:										*
 *															*
\************************************************************/


#include <hip/hip_runtime.h>
#include<stdio.h>

int main() {
	int count;
	int device;
	hipDeviceProp_t GPU;

	printf("Hola Mundo Cuda\n");
	hipGetDeviceCount(&count);

	printf("total GPGPUS: %d\n",count);

	device = (count==1) ? 0:-1;

	hipGetDeviceProperties(& GPU, device);
	printf("GPU name: %s \
		\n GPU global memory(GB) %u \
		\n GPU shared memory(KB) %u \
		\n GPU warpsize %d \
		\n GPU maxThreadsPerBlock %d \
		\n GPU multiprocesadores %d \
	       	\n"\
		, GPU.name, (unsigned int)(GPU.totalGlobalMem/(1024*1000*1000))\
		, (unsigned int) GPU.sharedMemPerBlock/1024 ,GPU.warpSize\
		, GPU.maxThreadsPerBlock, GPU.multiProcessorCount);

	return 0;
}
