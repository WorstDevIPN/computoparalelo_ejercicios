/********************************************************
 *														*
 * Autor: Programacion_en_CUDA-Moises Arreola Zamora	*
 *														*
 * Fecha:12/10/2020										*
 * 														*
 * Lanzamiento de Kernel								*
 *														*
 * 														*
 * compilacion con: 									*
 * nvcc fractal.cu -o fractal.out -lglut -lGL -lGLU		*
*********************************************************/
//librerias

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

//definiciones
#define NUM_MATRICES	3
#define PROFUNDIDAD		24

#define PRIMER_SEG		24
#define	SEGUNDO_SEG		48
#define TERCER_SEG		72


__global__ void rellenado( int *array) {
	int id_global 	= threadIdx.x + blockDim.x * blockIdx.x;
	int id_thread 	= threadIdx.x;
	int id_block 	= blockIdx.x;
	//no usar for, apoyarse del no de bloques o el no de threads	
	array[id_global] = id_thread;
	array[id_global+24] = id_block;
	array[id_global+48] = id_global;
	
}

// MAIN:
int main(int argc, char** argv)
{
// declaracion var host
int i;
int bloques = 0;
int hilos 	= 0;
int *host_x;

//var device
int *dev_array;

//reserva al host
host_x = (int*)malloc(NUM_MATRICES*PROFUNDIDAD*sizeof(int));

// reserva en el device
hipMalloc( (void**)&dev_array, NUM_MATRICES*PROFUNDIDAD*sizeof(int) );

//ingreso de valores 
printf("Ejecutando Practica 4\n");

//invocacion kernel
for (i = 0; i < NUM_MATRICES; i++){
	if(i == 0){
		bloques	=	1;
		hilos	=	24;
		rellenado<<<1,24>>>(dev_array);
	}
	else if( i == 1){
		bloques	=	24;
		hilos	=	1;
		rellenado<<<bloques,hilos>>>(dev_array);
	}
	else {
		bloques	=	4;
		hilos	=	6;
		rellenado<<<bloques,hilos>>>(dev_array);
	}
	printf("\nbloques %d hilos %d\n", bloques,hilos);
	//copia de device a host
	hipMemcpy(host_x, dev_array, NUM_MATRICES*PROFUNDIDAD*sizeof(float),hipMemcpyDeviceToHost);
	printf("hilo\n");
	for(int i = 0; i<PROFUNDIDAD; i++){
		printf("%d \t",host_x[i]);
	}
	printf("\nbloque\n");
	for(int i = 0; i<PROFUNDIDAD; i++){
		printf("%d \t",host_x[i+24]);
	}
	printf("\ngloabal\n");
	for(int i = 0; i<PROFUNDIDAD; i++){
		printf("%d \t",host_x[i+48]);
	}

}

hipFree(dev_array);
//~ free(host_x);
printf("\nFin de Programa \n");

fflush(stdin);
return 0;
}

