/************************************************************\
 *															*
 * Titulo			:	Ejecucion de hilos					*
 * Organizacion		:	CIC-IPN								*
 * Autor	       	:	Oswaldo Franco						*
 * e-mail			:	oswaldo1509@gmail.com				*
 * Periodo			:	Semestre B20						*
 * Dependencies		:										*
 *															*
\************************************************************/


#include <hip/hip_runtime.h>
#include<stdio.h>

#define N 24

__global__ void kernel(int *thread, int *block, int *global){
	int myId = threadIdx.x;
	int myBlock = blockIdx.x;
	int myGlobal = threadIdx.x + blockDim.x * blockIdx.x;

	thread [myGlobal] = myId;
	block [myGlobal] = myBlock;
	global [myGlobal] = myGlobal;
}

int main(){
	int *hst_thread, *hst_block, *hst_global;
	int *dev_thread, *dev_block, *dev_global;
	int option;

	hst_thread = (int*) malloc(N*sizeof(int));
	hst_block = (int*) malloc(N*sizeof(int));
	hst_global = (int*) malloc(N*sizeof(int));

	hipMalloc( (void**)&dev_thread, N*sizeof(int) );
	hipMalloc( (void**)&dev_block, N*sizeof(int) );
	hipMalloc( (void**)&dev_global, N*sizeof(int) );
	printf("Ingresa opcion: \n\
		0: 1B24H\n\
		1: 24B1H\n\
		2: 4B6H\n");
	scanf("%d",&option);
	printf("Opcion %d:", option);
	switch (option){
		case 1:
			kernel<<<N,1>>>(dev_thread, dev_block, dev_global);
			printf("24 bloques 1 hilo \n");
			break;
		case 2:
			kernel<<<4,6>>>(dev_thread, dev_block, dev_global);
			printf("4 bloques 6 hilos \n");
			break;
		default:
			kernel<<<1,N>>>(dev_thread, dev_block, dev_global);
			printf("1 bloque 24 hilos \n");
			break;
	}

	hipMemcpy(hst_thread, dev_thread, N*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(hst_block, dev_block, N*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(hst_global, dev_global, N*sizeof(int), hipMemcpyDeviceToHost);

	printf("indice de hilo\n");
	for (int i=0; i<N; i++)	printf("%d\t", hst_thread[i]);
	printf("\n");
	printf("indice de bloque\n");
	for (int i=0; i<N; i++)	printf("%d\t", hst_block[i]);
	printf("\n");
	printf("indice global\n");
	for (int i=0; i<N; i++)	printf("%d\t", hst_global[i]);
	printf("\n");

	hipFree(dev_thread);
	hipFree(dev_block);
	hipFree(dev_global);

	return 0;
}
