/****************************************************************\
 *								*
 * Titulo		:	Lanzamiento de kernel		*
 * Organizacion		:	CIC-IPN				*
 * Autor	       	:	Oswaldo Franco			*
 * e-mail		:	oswaldo1509@gmail.com		*
 * Periodo		:	Semestre B20			*
 * Dependencies		:					*
 *								*
\****************************************************************/


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

struct hipComplex {
        float   r;
        float   i;
        hipComplex( float a, float b ) : r(a), i(b) {}
};


__global__ void chicharronera( float a, float b, float c, hipComplex *x){
	// calculo de chicharronera
	// falta manejo de imaginarios a + bi
	float det;
	det = b*b - 4*a*c;
	if (det >= 0){
		x[0].r = (-b/2*a) + (sqrtf(det) / 2*a);
		x[0].i = 0.0;
		x[1].r = (-b/2*a) - (sqrtf(det) / 2*a);
		x[1].i = 0.0;
	}
	else{
		x[0].r = (-b/2*a);
		x[0].i = (sqrtf(-det) / 2*a);
                x[1].r = (-b/2*a);
		x[1].i = (sqrtf(-det) / 2*a);
	}
}

int main(){
	int a, b, c;
	hipComplex *hst_x;
	hipComplex *dev_x;

	// reserva de memoria en host y device para resultados
	hst_x = (hipComplex*)malloc( 2*sizeof(hipComplex));
	hipMalloc( (void**)&dev_x, 2*sizeof(hipComplex));
	
	// ingreso de datos
	printf("Ingresa los coeficientes a, b y c de la ecuacion ax^2 + bx + c = 0\n");
	scanf("%d %d %d", &a, &b, &c);

	// lanzamiento de kernel
	chicharronera<<<1,1>>>((float)a,(float)b,(float)c,dev_x);

	// recuperar resultados de Device a Host
	hipMemcpy(hst_x, dev_x, 2*sizeof(hipComplex), hipMemcpyDeviceToHost);

	//  mostrar resultados
	printf("\nx1 = %.2f + %.2fi \t x2 = %.2f - %.2fi\n", hst_x[0].r, hst_x[0].i, hst_x[1].r, hst_x[1].i);

	// liberar memoria en host y device
	hipFree(dev_x);
	free(hst_x);

	return 0;

}
