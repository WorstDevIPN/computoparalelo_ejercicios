#include "hip/hip_runtime.h"
#include "common.h"
#include<stdio.h>
#include<stdlib.h>

#define N 32

__global__ void merge(int *left, int *right, int *merged, int size)	{

	int i = 0, j = 0, k = 0;

	while ( i<size/2 && j<size/2 ) {
		if ( left[i] <= right[j] ) {
			merged[k] = left[i];
			i ++;
		}
		else {
			merged[k] = right[j];
			j ++;
		}
		k++;
	}
	while ( i<size/2 ) {
		merged[k] = left[i];
		i ++;
		k ++;
	}
	while ( j<size/2 ){
		merged[k] = right[j];
		j ++;
		k ++;
	}

}


__global__ void mergesort (int *LEFT, int *RIGHT,  int size)	{
	int tid = threadIdx.x;
	int dsize = size >> 1;

	if ( size == 1) {
		return;
	}
	else {	
	        int *left;
        	int *right;
			hipMalloc((void**)&left, sizeof(int) * dsize);
			hipMalloc((void**)&right, sizeof(int) * dsize);
		if (tid == 0){                
			memcpy(left, LEFT, dsize*sizeof(int));
			memcpy(right, &LEFT[dsize], dsize*sizeof(int));
			mergesort<<<1,2>>>(left, right, dsize);
			merge<<<1,1>>>(left, right, LEFT, size);
		}
		else if (tid == 1){
			memcpy(left, RIGHT, dsize*sizeof(int));
			memcpy(right, &RIGHT[dsize], dsize*sizeof(int));
			mergesort<<<1,2>>>(left, right, dsize);
			merge<<<1,1>>>(left, right, RIGHT, size);
		}
	}
	__syncthreads();
}


int main (void)	{
	int *l;
	int *r;
	int *m;
	int *dev_l, *dev_r, *dev_m;
	int hi=2*N, lo=0;
    double iStart, iElaps;
    
    
   	l = (int*)malloc(sizeof(int) * N);
	r = (int*)malloc(sizeof(int) * N);
	m = (int*)malloc(sizeof(int) * 2 * N);
    
	srand(time(NULL));

	hipMalloc( (void**)&dev_l, N*sizeof(int) ) ;
	hipMalloc( (void**)&dev_r, N*sizeof(int) );
	hipMalloc( (void**)&dev_m, 2*N*sizeof(int) );

	for(int i = 0; i < N; i++) {
        	l[i] = rand() % (hi - lo) + 1;
	        r[i] = rand() % (hi - lo) + 1;
	}	

	printf("Data to merge: {");
	for (int i=0; i<N; i++)	{
		printf("%d,", l[i]);
	}
	printf("},{");
	for (int i=0; i<N; i++)	{
		printf("%d,", r[i]);
	}
	printf("}\n");


	hipMemcpy( dev_l, l, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( dev_r, r, N*sizeof(int), hipMemcpyHostToDevice);
	
	iStart = seconds();

	mergesort<<<1,2>>>(dev_l, dev_r, N);

    hipMemcpy( l, dev_l, N*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy( r, dev_r, N*sizeof(int),hipMemcpyDeviceToHost);

	merge<<<1,1>>>(dev_l, dev_r, dev_m, 2*N);
	
	iElaps = seconds() - iStart;
	
	hipMemcpy( m, dev_m, 2*N*sizeof(int),hipMemcpyDeviceToHost);

	printf("Merged Data: {");
	for (int i=0; i<N; i++)	{
		printf("%d,%d,", m[2*i], m[2*i+1]);
	}
	printf("}\n");
	
	printf("KernelOnGPU Time elapsed %f sec\n",iElaps);
	
	free(l);
	free(r);
	free(m);

	hipFree( dev_l );
	hipFree( dev_r );
	hipFree( dev_m );

	hipDeviceSynchronize();
}

