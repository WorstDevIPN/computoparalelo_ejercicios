#include "hip/hip_runtime.h"
/********************************************************
 *														*
 * Autor: Programacion_en_CUDA-Moises Arreola Zamora	*
 *														*
 * Fecha:19/10/2020										*
 * 														*
 * Lanzamiento de Kernel con multiples hilos y bloques:	*
 * los hilos por bloque son fijos a 4 y el numero de 	*
 * bloques depende del tamaño del arreglo de elementos	*
 * Cada hilo puede invertir como maximo un par de 		*
 * elementos											*
 * 														*
 * compilacion con: 									*
 * nvcc practica_5.cu -o practica_5.o				 	*
*********************************************************/
//librerias

#include <stdio.h>
#include <stdlib.h>
#include ""
#include <hip/hip_runtime.h>

//definiciones
#define NUM_HILOS	4
#define TAM_MIN		3


__global__ void reordena(int N, int punto_medio, int bandera, int *arreglo_ppal) {
	int id_global 	= threadIdx.x + (blockDim.x * blockIdx.x);
	int aux_swap;
	//~ printf("tamaño = %d",N);
	if (N <= TAM_MIN){
		//~ printf("Arreglo de tamaño minimo\n");
		if (id_global == 0){
			if (N == 3){
				aux_swap 		= arreglo_ppal[0];
				arreglo_ppal[0] = arreglo_ppal[2];
				arreglo_ppal[2]	= aux_swap;
				return;
			}
			else if(N == 2){
				aux_swap 		= arreglo_ppal[0];
				arreglo_ppal[0] = arreglo_ppal[1];
				arreglo_ppal[1] = aux_swap;
				return;
			}
			else{
				printf("Tamaño invalido\n");
				return;
			}
		}
		else
			return;
	}
	else{
		if ( (id_global + 1) == punto_medio && bandera > 0){
			//~ printf("Se llego a punto medio y se termina el swap\n");
			return;
		} 
		else {
			aux_swap 					= arreglo_ppal[id_global];
			arreglo_ppal[id_global] 	= arreglo_ppal[N - (id_global+1)];
			arreglo_ppal[N - (id_global+1)] 	= aux_swap;
		}
	}
}

__host__ void check_CUDA_Error(const char *mensaje){
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess){
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error),
		mensaje );
		printf("\npulsa INTRO para finalizar...");
		fflush(stdin);
		char tecla =getchar();
		exit(-1);
	}
}


// MAIN:
int main(int argc, char** argv)
{
	// declaracion var host
	int i;
	int tam_vector;
	int bloques = 0; 		//dependiente del tamaño del arreglo
		
	int *host_arreglo_ordenado;
	int *host_arreglo_reordenado;
	
	//Temporizacion
	//~ hipDeviceProp_t myGPU;
	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	//var device
	int *dev_arreglo_ordenado;
	
	//ingreso de valores 
	printf("Ejecutando Practica 5\t ");
	printf("Ingresa el tamaño del Vector \n");
	scanf("%d",&tam_vector);
	if(tam_vector <= 1){
		printf("El tamaño es invalido, saliendo del programa \n");
		return 0;
	}
	
	//variable de control
	bloques 	= tam_vector/4;
	if(bloques < 1) bloques = 1;
	int media 	= tam_vector/2; 	//limite para habilitar hilos
	int modulo	= tam_vector%2;
	
	//~ printf("Los valores de las variables de control:\nNo. de bloques: %d\t Punto medio: %d\t Modulo: %d \n",
			//~ bloques, media, modulo);

	//reserva al host
	host_arreglo_ordenado = (int*)malloc(tam_vector*sizeof(int));
	host_arreglo_reordenado = (int*)malloc(tam_vector*sizeof(int));

	// reserva en el device
	hipMalloc( (void**)&dev_arreglo_ordenado, tam_vector*sizeof(int) );
	check_CUDA_Error("Error en hipMalloc\n");
	
	//rellenado del arreglo
	printf("\nLos valores del arreglo son: \n");
	srand ( (int)time(NULL) );
	
	for (i=0; i < tam_vector; i++){
		host_arreglo_ordenado[i] = (int)( rand() % 10 );
		printf("Arreglo_Ordenado[%d] = %d \n", i,
				host_arreglo_ordenado[i] );
	}
	// copia de datos
	hipMemcpy(dev_arreglo_ordenado, host_arreglo_ordenado, tam_vector*sizeof(int), 
		hipMemcpyHostToDevice);
	check_CUDA_Error("Error en hipMemcpy H2D\n");

	//invocacion kernel
	hipEventRecord(start,0);
	reordena<<<bloques,NUM_HILOS>>>(tam_vector, media, modulo, dev_arreglo_ordenado);
	check_CUDA_Error("Error en reordena\n");
	
	//copia de device a host
	//~ hipDeviceSynchronize();
	hipMemcpy(host_arreglo_reordenado, dev_arreglo_ordenado, tam_vector*sizeof(int),
		hipMemcpyDeviceToHost);
	check_CUDA_Error("Error en hipMemcpy D2H\n");
	
	//parando cronometro
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	
	//~ // salida	
	//~ desplegar_datos(host_arreglo_reordenado);
	printf("\nLos valores del arreglo reordenados son:\n");
	for (i=0; i < tam_vector; i++){
		printf("Arreglo_ReOrdenado[%d] = %d \n", i,
				host_arreglo_reordenado[i] );
	}
	
	printf("\nTiempo transcurrido en Device : %.3f\n",elapsedTime);
	//memory dump
	hipFree(dev_arreglo_ordenado);
	free(host_arreglo_ordenado);
	free(host_arreglo_reordenado);
	
	printf("\nFin de Programa \n");

	fflush(stdin);
	return 0;
}

