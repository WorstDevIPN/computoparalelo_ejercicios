#include "hip/hip_runtime.h"
/********************************************************
 *														*
 * Autor: Programacion_en_CUDA-Moises Arreola Zamora	*
 *														*
 * Fecha:21/10/2020										*
 * 														*
 * Lanzamiento de Kernel con multiples hilos y bloques:	*
 * 														*
 * Algoritmo de ordenamiento mergesort 					*
 * 														*
 * compilacion con (revisar por guiones):				*
 * nvcc -rdc=true −arch=compute_35 mergesort.cu −o sort	*
 * nvprof --unified-memory-profiling off ./sort			*
*********************************************************/
//librerias

#include <stdio.h>
#include <stdlib.h>
#include ""
//~ #include "../include/common/book.h"
#include <hip/hip_runtime.h>

//definiciones
#define N 	16 //

__global__ void merge(float *lado_izq, float *lado_der, float *unido, int sizes) {
	int i = 0;
	int j = 0;
	int k = 0;

	while ( i<sizes/2 && j<sizes/2 ){
		if ( lado_izq[i] <= lado_der[j] ){
			unido[k] = lado_izq[i];
			i ++;
		}
		else{
			unido[k] = lado_der[j];
			j ++;
		}
		k++;
	}
	while ( i<sizes/2 ){
		unido[k] = lado_izq[i];
		i ++;
		k ++;
	}
	while ( j<sizes/2 ){
		unido[k] = lado_der[j];
		j ++;
		k ++;
	}	
}

__global__ void merge_sort(float *L, float *R, int size){
	//~ int id_threadx 	= threadIdx.x ;
	//Obtencion del tamaño de arreglos
	int i;
	int tamano = size /2;
	
	//~ printf("El id del thread es %d : %d\n",id_threadx, blockDim.x);
	if (size == 1){
		//~ printf("Condicion de salida de recursion\n");
		return;
	}
	else{
		float *Lft;
		float *Rgt;
		
		hipMalloc((void**)&Lft, sizeof(float)*tamano);
		hipMalloc((void**)&Rgt, sizeof(float)*tamano);
		
		if (threadIdx.x == 0){			
			//intercambio de posiciones
			memcpy(Lft, L, tamano*sizeof(float));
			memcpy(Rgt, &L[tamano], tamano*sizeof(float));
			//recursion
			merge_sort<<<1,2>>>(Lft, Rgt, tamano);
			merge<<<1,1>>>(Lft, Rgt, L, size);		
		}
		else if(threadIdx.x == 1){
			//intercambio de posiciones
			memcpy(Lft, R, tamano*sizeof(float));
			memcpy(Rgt, &R[tamano], tamano*sizeof(float));
			//recursion
			merge_sort<<<1,2>>>(Lft, Rgt, tamano);
			merge<<<1,1>>>(Lft, Rgt, R, size);		
		}
	}
	__syncthreads();
}

__host__ void check_CUDA_Error(const char *mensaje){
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess){
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error),
		mensaje );
		printf("\npulsa INTRO para finalizar...");
		fflush(stdin);
		char tecla =getchar();
		exit(-1);
	}
}


// MAIN:
int main(int argc, char** argv)
{
	// declaracion var host
	int i;

	float *host_array;
	float *host_array_L;
	float *host_array_R;
	
	//Temporizacion
	//~ hipDeviceProp_t myGPU;
	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	//var device
	float *dev_array;
	float *dev_array_L;
	float *dev_array_R;
	
	//ingreso de valores 
		
	//variables de control
		
	//reserva al host
	host_array	 = (float*)malloc(N*sizeof(float));
	host_array_L = (float*)malloc(N/2*sizeof(float));
	host_array_R = (float*)malloc(N/2*sizeof(float));

	// reserva en el device
	hipMalloc( (void**)&dev_array, N*sizeof(float) );
	hipMalloc( (void**)&dev_array_L, N/2*sizeof(float) );
	hipMalloc( (void**)&dev_array_R, N/2*sizeof(float) );//*2
	check_CUDA_Error("Error en hipMalloc\n");
	
	//rellenado de la matriz
	printf("\nLos valores del arreglo son: \n");
	srand ( (float)time(NULL) );
	
	for (i=0; i < N; i++){
		host_array[i] = (float)( rand() %N+1);
		printf("Arreglo_Desordenado[%d] = %.0f \n", i,host_array[i] );
	}
	
	//rellenado de las matrices  izq y der
	for (i=0; i < N/2; i++){
		host_array_L[i] = host_array[i];
		host_array_R[i] = host_array[N/2+i];		
	}
	printf("Valores a ordenar Derecha:\n");
	for (i=0; i < N/2; i++){
		printf("D[%d]:%.0f\t",i,host_array_R[i]);	
	}
	printf("\nValores a ordenar Izquierda:\n");
	for (i=0; i < N/2; i++){
		printf("I[%d]:%.0f\t",i,host_array_L[i]);	
	}

	// copia de datos a device
	hipMemcpy(dev_array_L, host_array_L, N/2*sizeof(float), 
		hipMemcpyHostToDevice);
	hipMemcpy(dev_array_R, host_array_R, N/2*sizeof(float), 
		hipMemcpyHostToDevice);
	check_CUDA_Error("Error en hipMemcpy H2D\n");
	
	hipEventRecord(start,0);
	//invocacion kernel
	merge_sort<<<1,2>>>(dev_array_L, dev_array_R, N/2); // /2
	//respaldo	
	hipMemcpy(host_array_L, dev_array_L, N/2*sizeof(float),
		hipMemcpyDeviceToHost);
	hipMemcpy(host_array_R, dev_array_R, N/2*sizeof(float),
		hipMemcpyDeviceToHost);
	//
	merge<<<1,1>>>(dev_array_L, dev_array_R, dev_array, N);//*2 
	check_CUDA_Error("Error en modificar_matriz\n");
	
	//copia de device a host
	//~ hipDeviceSynchronize();
	//~ hipMemcpy(host_array_L, dev_array_L, N*sizeof(float),
		//~ hipMemcpyDeviceToHost);
	//~ hipMemcpy(host_array_R, dev_array_R, N*sizeof(float),
		//~ hipMemcpyDeviceToHost);
	hipMemcpy(host_array, dev_array, N*sizeof(float),
		hipMemcpyDeviceToHost);
	check_CUDA_Error("Error en hipMemcpy D2H\n");
	
	//parando cronometro
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	
	//~ // salida	
	//~ desplegar_datos(host_arreglo_reordenado);
	printf("\nLos valores del arreglo reordenados son:\n");
	for (i=0; i < N; i++){
		printf("Arreglo_ordenado[%d] = %.0f \n", i,
					host_array[i] );
	}
	
	printf("\nTiempo transcurrido en Device : %.3f\n",elapsedTime);
	//memory dump
	hipFree(dev_array_L);
	hipFree(dev_array_R);
	hipFree(dev_array);
	
	free(host_array);
	free(host_array_L);
	free(host_array_R);
	
	printf("\nFin de Programa \n");

	fflush(stdin);
	return 0;
}

