/****************************************************************\
 *																*
 * Titulo			:	Ejecucion de kernel						*
 * Organizacion		:	CIC-IPN									*
 * Autor	       	:	Oswaldo Franco							*
 * e-mail			:	oswaldo1509@gmail.com					*
 * Periodo			:	Semestre B20							*
 * Dependencies		:	cudaError.h								*
 * Referencia		:	Introduccion a la programacion			*
 *						en CUDA ( Universidad de Burgos )		*
 *																*
\****************************************************************/

// includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// declaracion de funciones
// GLOBAL: funcion llamada desde el host y ejecutada en el device (kernel)
__global__ void suma_GPU(int a, int b, int *c) {
	*c = a + b;
}
// HOST: funcion llamada y ejecutada desde el host
__host__ int suma_CPU(int a, int b) {
	return (a + b);
}

// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv) {
	// declaraciones
	int n1 = 1, n2 = 2, c = 0;
	int *hst_c;
	int m1 = 10, m2 = 20;
	int *dev_c;
	//size_t memLibre, memTotal;
	// reserva en el host
	hst_c = (int*)malloc( sizeof(int) );
	// reserva en el device
	hipMalloc( (void**)&dev_c, sizeof(int) );
	// llamada a la funcion suma_CPU
	c = suma_CPU(n1, n2);
	// resultados CPU
	printf("CPU:\n");
	printf("%2d + %2d = %2d \n",n1, n2, c);
	// llamada a la funcion suma_GPU
	suma_GPU<<<1,1>>>(m1, m2, dev_c);
	// recogida de datos desde el device
	hipMemcpy( hst_c, dev_c, sizeof(int), hipMemcpyDeviceToHost );
	// resultados GPU
	//cudaMemGetInfo(&memLibre, &memTotal);
	//printf("Libre %zu Total %zu\n",memLibre, memTotal);	
	printf("GPU:\n");
	printf("%2d + %2d = %2d \n",m1, m2, *hst_c);

	return 0;
}
