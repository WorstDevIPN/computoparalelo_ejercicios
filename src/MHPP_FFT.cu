#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "complex.h"

#define N 4

hipComplex * fft(hipComplex *, int);

int main(){
	hipComplex *hst_array;
	hst_array = (hipComplex*)malloc(N*sizeof(hipComplex));

	srand(time(NULL));

	for(int i=0; i<N; i++){
		hst_array[i] = hipComplex((float)i+1.0,0.0);
		printf("array %d = (%.2f,%.2fi)\n",i, hst_array[i].r, hst_array[i].i);
	}

	hipComplex *hst_fft = fft(hst_array, N);

	for(int i=0; i<N; i++){
		printf("array %d = (%.2f,%.2fi)\n",i, hst_fft[i].r, hst_fft[i].i);
	}
	
	return 0;
}

    /**
     * Performs the FFT to an array of complex values.
     * @param x The Complex[] array of values.
     * @return Complex[] representing the FFT of x.
     */


hipComplex * fft(hipComplex *x, int n) {
        if (n == 1) {
		return x;
        }
        // Even terms
        int len = (n % 2 != 0) ? (n + 1) / 2 : n / 2;

        //Complex[] even = new Complex[len];
	hipComplex *even;
	even = (hipComplex*)malloc(len*sizeof(hipComplex));

        for (int k = 0; k < len; k++) {
            even[k] = x[2 * k];
        }
        //Complex[] e = fft(even);
	hipComplex *e = fft(even, len);
	
        
        // Odd terms
        len = (n % 2 != 0) ? (n - 1) / 2 : n / 2;
        //Complex[] odd = new Complex[len];
	hipComplex *odd;
	odd = (hipComplex*)malloc(len*sizeof(hipComplex));
        for (int k = 0; k < len; k++) {
            odd[k] = x[2 * k + 1];
        }
        //Complex[] o = fft(odd);
	hipComplex *o = fft(odd, len);
        
        // Combine
        len = (n % 2 != 0) ? (n - 1) / 2 : n / 2;
        //Complex[] y = new Complex[n];
	
	hipComplex *y;
	y = (hipComplex*)malloc(len*sizeof(hipComplex));

        for (int k = 0; k < n/2; k ++) {
		double kth = -2 * k * M_PI / n;
		hipComplex wk = hipComplex(cos(kth), sin(kth));
		y[k] = e[k] + (wk*o[k]);
		y[k + n/2] = e[k]-(wk*o[k]);
        }

        return y;
}
  

hipComplex* ifft(hipComplex *x, int n) {
	hipComplex *y;
	y = (hipComplex*)malloc(n*sizeof(hipComplex));

	// Take conjugate
	for (int i = 0; i < n; i++) {
		y[i] = x[i].conjugate();
	}
        
	// Compute forward FFT
	y = fft(y, n);
        
	// Take conjugate again
	for (int i = 0; i < n; i++) {
		y[i] = y[i].conjugate();
	}
        
	// Divide by n
	for (int i = 0; i < n; i++) {
		y[i] = y[i].scale(1.0 / n);
	}
	return y;
}
