/****************************************************************\
 *																*
 * Titulo			:	Uso de memoria constante				*
 * Organizacion		:	CIC-IPN									*
 * Autor	       	:	Oswaldo Franco							*
 * e-mail			:	oswaldo1509@gmail.com					*
 * Periodo			:	Semestre B20							*
 * Dependencies		:	cudaError.h								*
 * Referencia		:	Introduccion a la programacion			*
 *						en CUDA ( Universidad de Burgos )		*
 *																*
\****************************************************************/

// includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// defines
#define N 8

// CUDA constants
__constant__ float dev_A[N][N];

// declaracion de funciones

// GLOBAL: funcion llamada desde el host y ejecutada en el device (kernel)
__global__ void traspuesta( float *dev_B) {
	// kernel lanzado con un solo bloque y NxN hilos
	int columna = threadIdx.x;
	int fila = threadIdx.y;
	int pos = columna + N*fila;
	// cada hilo coloca un elemento de la matriz final
	dev_B[pos] = dev_A[columna][fila];
}

// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv) {
	// declaraciones
	float *hst_A, *hst_B;;
	float *dev_B;
	
	// reserva en el host
	hst_A = (float*)malloc(N*N*sizeof(float));
	hst_B = (float*)malloc(N*N*sizeof(float));

	// reserva en el device
	hipMalloc( (void**)&dev_B, N*N*sizeof(float));
	// inicializacion
	for(int i=0; i<N*N; i++) {
		hst_A[i]= (float)i;
	}// copia de datos
	hipMemcpyToSymbol( HIP_SYMBOL(dev_A), hst_A, N*N*sizeof(float));

	// dimensiones del kernel
	dim3 Nbloques(1);

	dim3 hilosB(N,N);

	// llamada al kernel bidimensional de NxN hilos
	traspuesta<<<Nbloques,hilosB>>>(dev_B);

	// recogida de datos
	hipMemcpy( hst_B, dev_B, N*N*sizeof(float), hipMemcpyDeviceToHost );

	// impresion de resultados

	printf("Resultado:\n");
	printf("ORIGINAL:\n");
	for(int i=0; i<N; i++) {
		for(int j=0; j<N;j++) {
			printf("%2.0f ",hst_A[j + i*N]);
		}
		printf("\n");
	}
	
	printf("TRASPUESTA:\n");
	for(int i=0;i<N;i++) {
		for(int j=0; j<N; j++) {
			printf("%2.0f ",hst_B[j + i*N]);
		}
		printf("\n");
	}

	return 0;
}
