/********************************************************
 *														*
 * Autor: Programacion_en_CUDA-Moises Arreola Zamora	*
 *														*
 * Fecha:21/10/2020										*
 * 														*
 * Lanzamiento de Kernel con multiples hilos:			*
 * Calculo de la aproximacion de pi 					*
 *														*
 * compilacion con: 									*
 * nvcc practica_7.cu -o practica_7.o				 	*
*********************************************************/
//librerias

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>

//definiciones
#define NUM_HILOS	128

__global__ void calcular_pi_shared(float *matriz_ppal) {
	//reservado de memoria compartida de hilos
	__shared__ float array [NUM_HILOS];
	int id_threadx 	= threadIdx.x ;
	int hilo		= threadIdx.x +1;
	//calculo de la casilla y sincroniza
	array[id_threadx] = 1/(hilo*hilo);
	__syncthreads();
	//reduccion paralela para ir sumando terminos
	int salto = NUM_HILOS/2;
	while (salto){
		if(id_threadx < salto){
			array[id_threadx] = array[id_threadx] + array[id_threadx+salto];
		}
		__syncthreads();
		salto = salto>>1;
	}
	if (id_threadx == 0){
		//~ printf("\nEl valor aprox de pi es %f\n",sqrtf(array[id_threadx]*6));
		*matriz_ppal = array[id_threadx];
	}
}

__global__ void calcular_pi(float *matriz_ppal, float *arreglo) {
	//~ float array[NUM_HILOS];
	int id_threadx 	= threadIdx.x ;
	int hilo		= threadIdx.x +1;
	//calculo de la casilla y sincroniza
	arreglo[id_threadx] = 1/(hilo*hilo);
	__syncthreads();
	//reduccion paralela para ir sumando terminos
	int salto = NUM_HILOS/2;
	while (salto){
		if(id_threadx < salto){
			arreglo[id_threadx] = arreglo[id_threadx] + arreglo[id_threadx+salto];
		}
		__syncthreads();
		salto = salto/2;
	}
	if (id_threadx == 0){
		//~ printf("\nEl valor aprox de pi es %f\n",sqrtf(arreglo[id_threadx]*6));
		*matriz_ppal = arreglo[id_threadx];
	}
}

__host__ void check_CUDA_Error(const char *mensaje){
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess){
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error),
		mensaje );
		printf("\npulsa INTRO para finalizar...");
		fflush(stdin);
		char tecla =getchar();
		exit(-1);
	}
}


// MAIN:
int main(int argc, char** argv)
{
	// declaracion var host
	float host_resultado;
	
	//Temporizacion
	//~ cudaDeviceProp myGPU;
	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	//var device
	float *dev_matriz;
	float *dev_matriz_global;

	hipMalloc( (void**)&dev_matriz, sizeof(float) );
	hipMalloc( (void**)&dev_matriz_global, NUM_HILOS*sizeof(float) );
	check_CUDA_Error("Error en cudaMalloc\n");
	
	//invocacion kernel
	hipEventRecord(start,0);
	
	calcular_pi_shared<<<1,NUM_HILOS>>>(dev_matriz);
	check_CUDA_Error("Error en calcular_pi\n");
		//copia de device a host
	hipMemcpy(&host_resultado, dev_matriz, sizeof(float),
		hipMemcpyDeviceToHost);
	check_CUDA_Error("Error en hipMemcpy D2H\n");
	//~ // salida	
	printf("\nLa aproximacion de Pi por memoria compartida es: %.9f\n",sqrt(6*host_resultado));
	
	hipFree(dev_matriz);

	hipMalloc( (void**)&dev_matriz, sizeof(float) );
	check_CUDA_Error("Error en cudaMalloc\n");
	
	calcular_pi<<<1,NUM_HILOS>>>(dev_matriz,dev_matriz_global);
	check_CUDA_Error("Error en calcular_pi\n");
		//copia de device a host
	hipMemcpy(&host_resultado, dev_matriz, sizeof(float),
		hipMemcpyDeviceToHost);
	check_CUDA_Error("Error en hipMemcpy D2H\n");
	//~ // salida	
	printf("\nLa aproximacion de Pi por memoria global es: %.9f\n",sqrt(6*host_resultado));
	
	
	//parando cronometro
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	
	
	
	printf("\nTiempo transcurrido en Device : %.3f\n",elapsedTime);
	//memory dump
	hipFree(dev_matriz);
	hipFree(dev_matriz);
	hipFree(dev_matriz_global);
	//~ cudaFree(dev_matriz_ret);
	//~ free(host_resultado);
	//~ free(host_matriz_modificada);
	
	printf("\nFin de Programa \n");

	fflush(stdin);
	return 0;
}

