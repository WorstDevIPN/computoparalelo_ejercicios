/****************************************************************\
 *																*
 * Titulo			:	Ejecucion de hilos						*
 * Organizacion		:	CIC-IPN									*
 * Autor	       	:	Oswaldo Franco							*
 * e-mail			:	oswaldo1509@gmail.com					*
 * Periodo			:	Semestre B20							*
 * Dependencies		:	cudaError.h								*
 * Referencia		:	Introduccion a la programacion			*
 *						en CUDA ( Universidad de Burgos )		*
 *																*
\****************************************************************/

// includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// defines
#define N 16

// tamano de los vectores
#define BLOCK 5 // tamano del bloque

// declaracion de funciones
// GLOBAL: funcion llamada desde el host y ejecutada en el device (kernel)
__global__ void suma( float *a, float *b, float *c ) {
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	// Solo trabajan N hilos
	if (myID < N) {
		c[myID] = a[myID] + b[myID];
	}
}

// MAIN: rutina principal ejecutada en el host

int main(int argc, char** argv) {
	// declaraciones
	float *vector1, *vector2, *resultado;
	float *dev_vector1, *dev_vector2, *dev_resultado;
	
	// reserva en el host
	vector1 = (float *)malloc(N*sizeof(float));
	vector2 = (float *)malloc(N*sizeof(float));
	resultado = (float *)malloc(N*sizeof(float));
	
	// reserva en el device
	hipMalloc( (void**)&dev_vector1, N*sizeof(float));
	hipMalloc( (void**)&dev_vector2, N*sizeof(float));
	hipMalloc( (void**)&dev_resultado, N*sizeof(float));
	
	// inicializacion de vectores
	for (int i = 0; i < N; i++) {
		vector1[i] = (float) rand() / RAND_MAX;
		vector2[i] = (float) rand() / RAND_MAX;
	}
	
	// copia de datos hacia el device
	hipMemcpy(dev_vector1, vector1, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_vector2, vector2, N*sizeof(float), hipMemcpyHostToDevice);
	
	// lanzamiento del kernel
	// calculamos el numero de bloques necesario para un tamaño de bloque fijo
	int nBloques = N/BLOCK;
	if (N%BLOCK != 0) {
		nBloques = nBloques + 1;
	}

	int hilosB = BLOCK;
	printf("Vector de %d elementos\n", N);
	printf("Lanzamiento con %d bloques (%d hilos)\n", nBloques, nBloques*hilosB);
	suma<<< nBloques, hilosB >>>( dev_vector1, dev_vector2, dev_resultado );

	// recogida de datos desde el device
	hipMemcpy(resultado, dev_resultado, N*sizeof(float), hipMemcpyDeviceToHost);
	// impresion de resultados
	printf( "> vector1:\n");
	for (int i = 0; i < N; i++) {
		printf("%.2f ", vector1[i]);
	}
	printf("\n");

	printf( "> vector2:\n");
	for (int i = 0; i < N; i++) {
		printf("%.2f ", vector2[i]);
	}
	printf("\n");

	printf( "> SUMA:\n");
	for (int i = 0; i < N; i++) {
		printf("%.2f ", resultado[i]);
	}
	printf("\n");
	
	// liberamos memoria en el device
	hipFree( dev_vector1 );
	hipFree( dev_vector2 );
	hipFree( dev_resultado );
	
	// salida
	//printf("\npulsa INTRO para finalizar...");
	return 0;
}
