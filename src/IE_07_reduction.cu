#include "hip/hip_runtime.h"
/****************************************************************\
 *																*
 * Titulo			:	Uso de memoria compartida				*
 * Organizacion		:	CIC-IPN									*
 * Autor	       	:	Oswaldo Franco							*
 * e-mail			:	oswaldo1509@gmail.com					*
 * Periodo			:	Semestre B20							*
 * Dependencies		:	hipError_t.h								*
 * Referencia		:	Introduccion a la programacion			*
 *						en CUDA ( Universidad de Burgos )		*
 *																*
\****************************************************************/

#include<stdio.h>
#include<stdlib.h>

#define N 16

__global__ void reduction( float *vector, float *suma){
	__shared__ float temporal[N];

	int myID = threadIdx.x;
	temporal[myID] = vector[myID];
	__syncthreads();

	int salto = N/2;

	while(salto){
		if(myID< salto){
			temporal[myID] = temporal[myID] + temporal[myID+salto];
		}
		__syncthreads();
		salto = salto/2;
	}

	if(myID==0)	*suma = temporal[myID];
}

int main(){

	float *hst_vector;
	float hst_suma;
	float *dev_vector;
	float *dev_suma;


	hst_vector = (float*)malloc(N*sizeof(float));


	hipMalloc( (void**)&dev_vector, N*sizeof(float));
	hipMalloc( (void**)&dev_suma, sizeof(float));

	srand(time(NULL));

	for(int i=0; i<N; i++)	hst_vector[i] = (float) (rand()%16);

	hipMemcpy( dev_vector, hst_vector, N*sizeof(float), hipMemcpyHostToDevice);

	reduction<<<1,N>>>(dev_vector, dev_suma);

	hipMemcpy( &hst_suma, dev_suma, sizeof(float), hipMemcpyDeviceToHost);

	for (int i=0; i<N; i++){
		printf("%.2f\t", hst_vector[i]);
	}
	printf("\n");

	printf("suma %.2f\n", hst_suma);
	hipFree(dev_vector);
	hipFree(dev_suma);

	return 0;
}
