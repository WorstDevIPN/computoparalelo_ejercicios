#include "hip/hip_runtime.h"
/********************************************************
 *														*
 * Autor: Programacion_en_CUDA-Moises Arreola Zamora	*
 *														*
 * Fecha:19/10/2020										*
 * 														*
 * Lanzamiento de Kernel con multiples hilos y bloques:	*
 * Manipulacion de los datos usando arreglos multidim	*
 * con el manejo de las variables dim3.					*
 * Realizar la sustitucion de valores de una matriz de 	*
 * 16 por 16 elementos manejando el mismo no de hilos	*
 * 														*
 * compilacion con: 									*
 * nvcc practica_6.cu -o practica_6.o				 	*
*********************************************************/
//librerias

#include <stdio.h>
#include <stdlib.h>
#include ""
#include <hip/hip_runtime.h>

//definiciones
#define NUM_HILOS	16
//~ #define TAM_MIN		3


__global__ void modificar_matriz(int *matriz_ppal, int *matriz_mod) {
	//~ int id_globalx 	= threadIdx.x + (blockDim.x * blockIdx.x);
	//~ int id_globaly 	= threadIdx.y + (blockDim.y * blockIdx.y);
	int id_threadx 	= threadIdx.x ;
	int id_thready 	= threadIdx.y ;
	printf("Hilo id_globalx[%d] id_globaly[%d] = %d\n",id_threadx, id_thready,matriz_ppal[id_threadx+id_thready*NUM_HILOS]);
	
	//discriminar bordes
	if (id_threadx == 0 || id_thready == 0 || id_threadx == NUM_HILOS-1 || id_thready == NUM_HILOS-1 ){
		matriz_mod[id_threadx+id_thready*NUM_HILOS] = matriz_ppal[id_threadx+id_thready*NUM_HILOS];
		return;
	}
	else{
	//vaceado de datos
		matriz_mod[id_threadx+id_thready*NUM_HILOS] = matriz_ppal[(id_threadx-1)+id_thready*NUM_HILOS] +
				matriz_ppal[id_threadx+(id_thready-1)*NUM_HILOS] + matriz_ppal[(id_threadx+1)+id_thready*NUM_HILOS] +
				matriz_ppal[id_threadx+(id_thready+1)*NUM_HILOS];
	}
}

__host__ void check_CUDA_Error(const char *mensaje){
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess){
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error),
		mensaje );
		printf("\npulsa INTRO para finalizar...");
		fflush(stdin);
		char tecla =getchar();
		exit(-1);
	}
}


// MAIN:
int main(int argc, char** argv)
{
	// declaracion var host
	int i;
	int contador = 1;

	int *host_matriz_original;
	int *host_matriz_modificada;
	
	//Temporizacion
	//~ hipDeviceProp_t myGPU;
	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	//var device
	int *dev_matriz;
	int *dev_matriz_ret;
	
	//ingreso de valores 
		
	//variables de control
		
	//reserva al host
	host_matriz_original = (int*)malloc(NUM_HILOS*NUM_HILOS*sizeof(int));
	host_matriz_modificada = (int*)malloc(NUM_HILOS*NUM_HILOS*sizeof(int));

	// reserva en el device
	hipMalloc( (void**)&dev_matriz, NUM_HILOS*NUM_HILOS*sizeof(int) );
	hipMalloc( (void**)&dev_matriz_ret, NUM_HILOS*NUM_HILOS*sizeof(int) );
	check_CUDA_Error("Error en hipMalloc\n");
	
	//rellenado de la matriz
	printf("\nLos valores del arreglo son: \n");
	srand ( (int)time(NULL) );
	
	for (i=0; i < NUM_HILOS*NUM_HILOS; i++){
		
		if (i == (contador * NUM_HILOS)) {
			printf("\n");
			contador+=1;
		}
		host_matriz_original[i] = (int)( rand() % 2 );
		printf("Matriz_Inicial[%d] = %d \n", i,
					host_matriz_original[i] );
	}
	contador = 1;
	// copia de datos
	hipMemcpy(dev_matriz, host_matriz_original, NUM_HILOS*NUM_HILOS*sizeof(int), 
		hipMemcpyHostToDevice);
	check_CUDA_Error("Error en hipMemcpy H2D\n");
	
	//dimensiones del kernel
	dim3 Nbloques(1);
	dim3 Nhilos(NUM_HILOS,NUM_HILOS); 	//16*16

	//invocacion kernel
	hipEventRecord(start,0);
	
	modificar_matriz<<<Nbloques,Nhilos>>>(dev_matriz, dev_matriz_ret);
	check_CUDA_Error("Error en modificar_matriz\n");
	
	//copia de device a host
	//~ hipDeviceSynchronize();
	hipMemcpy(host_matriz_modificada, dev_matriz_ret, NUM_HILOS*NUM_HILOS*sizeof(int),
		hipMemcpyDeviceToHost);
	check_CUDA_Error("Error en hipMemcpy D2H\n");
	
	//parando cronometro
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	
	//~ // salida	
	//~ desplegar_datos(host_arreglo_reordenado);
	printf("\nLos valores del arreglo reordenados son:\n");
	for (i=0; i < NUM_HILOS*NUM_HILOS; i++){
		if (i == (contador * NUM_HILOS)) {
			printf("\n");
			contador+=1;
		}
		printf("Matriz_Modificada[%d] = %d \n", i,
					host_matriz_modificada[i] );
	}
	
	printf("\nTiempo transcurrido en Device : %.3f\n",elapsedTime);
	//memory dump
	hipFree(dev_matriz);
	hipFree(dev_matriz_ret);
	free(host_matriz_original);
	free(host_matriz_modificada);
	
	printf("\nFin de Programa \n");

	fflush(stdin);
	return 0;
}

