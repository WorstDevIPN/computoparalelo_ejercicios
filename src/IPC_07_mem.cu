/****************************************************************\
 *																*
 * Titulo			:	Uso de memoria compartida				*
 * Organizacion		:	CIC-IPN									*
 * Autor	       	:	Oswaldo Franco							*
 * e-mail			:	oswaldo1509@gmail.com					*
 * Periodo			:	Semestre B20							*
 * Dependencies		:	cudaError.h								*
 *																*
\****************************************************************/


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#define N 32
__global__ void kernelshared( float *aprox_pi){
	__shared__ float vector[N];
	int myID = threadIdx.x;

	vector[myID] = 1.0/((myID + 1)*(myID + 1));
	__syncthreads();

	int salto = N/2;

	while(salto){
		if(myID<salto){
			vector[myID] = vector[myID] + vector[myID + salto];
		}
		__syncthreads();
		salto = salto >> 1;
	}
	if(myID==0) *aprox_pi = vector[myID];

}


__global__ void kernelglobal( float *vector, float *aprox_pi){

	int myID = threadIdx.x;
	vector[myID] = 1.0/((myID + 1)*(myID + 1));
	__syncthreads();

	int salto = N/2;

	while(salto){
		if(myID<salto){
			vector[myID] = vector[myID] + vector[myID + salto];
		}
		__syncthreads();
		salto = salto >> 1;
	}

	__syncthreads();

	if(myID==0) *aprox_pi = vector[myID];

}

int main(){

	float hst_aprox_pi;
	float *dev_aprox_pi;
	hipDeviceProp_t myGPU;
	hipEvent_t start, stop;
	hipEvent_t startG, stopG;
	float elapsedTime, elapsedTimeG;
	float *dev_vector;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&startG);
	hipEventCreate(&stopG);


	hipGetDeviceProperties( &myGPU, 0);
	printf("maxThreadPerBlock: %d\n", myGPU.maxThreadsPerBlock);
	printf("sharedMemPerBlock: %zu\n", myGPU.sharedMemPerBlock/1024);
	printf("Launched threads: %d\n", N);

	hipMalloc( (void**)&dev_aprox_pi, sizeof(float) );

	hipEventRecord(start,0);
	kernelshared<<<1,N>>>(dev_aprox_pi);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	
	hipMemcpy(&hst_aprox_pi, dev_aprox_pi, sizeof(float), hipMemcpyDeviceToHost);

	printf("Elapsed Time: %.3f\n",elapsedTime);
	printf("Valor de Pi %.8f\n", sqrt(hst_aprox_pi*6));

	hipFree(dev_aprox_pi);
	
	hipMalloc( (void**)&dev_aprox_pi, sizeof(float) );
	hipMalloc( (void**)&dev_vector, N*sizeof(float) );

	hipEventRecord(startG,0);
	kernelglobal<<<1,N>>>(dev_vector, dev_aprox_pi);
	hipEventRecord(stopG,0);
	hipEventSynchronize(stopG);
	hipEventElapsedTime(&elapsedTimeG, startG, stopG);

	hipMemcpy(&hst_aprox_pi, dev_aprox_pi, sizeof(float), hipMemcpyDeviceToHost);
	
	printf("Elapsed Time: %.3f\n",elapsedTimeG);
	printf("Valor de Pi %.8f\n", sqrt(hst_aprox_pi*6));
	
	hipFree(dev_aprox_pi);

	return 0;
}
