#include "hip/hip_runtime.h"
/********************************************************
 *														*
 * Autor: Programacion_en_CUDA-Moises Arreola Zamora	*
 *														*
 * Fecha:28/10/2020										*
 * 														*
 * Lanzamiento de Kernel con multiples hilos y bloques:	*
 * Generacion de imagenes por medio de bloques de 		*
 * multiples hilos (16x16).						 		*
 * Dibujar un tablero de ajedrez de dim de 8 x 8  		*
 * 														*
 * compilacion con: 									*
 * nvcc practica_9.cu -o practica_9.o -lglut -lGL -lGLU	*
*********************************************************/
//librerias

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "../include/common/cpu_bitmap.h"
#include ""


//definiciones
#define DIM 1024 // Dimensiones del Bitmap 1024

__global__ void kernel( unsigned char *imagen )
{
	//bloques
	// coordenada horizontal
	int bx = blockIdx.x;
	// coordenada vertical
	int by = blockIdx.y;
	//hilos
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	// coordenada vertical
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	// coordenada global de cada pixel
	int pixel = x + y * blockDim.x * gridDim.x;
	// cada bloque pinta un pixel con un color arbitrario
	if((bx+by)%256 == 0){
		imagen[pixel *4 + 0] = 0;// canal R
		imagen[pixel *4 + 1] = 0;// canal G
		imagen[pixel *4 + 2] = 0;// canal B
		imagen[pixel *4 + 3] = 255;								// canal alfa
	}
	else{
		imagen[pixel *4 + 0] = 255;// canal R
		imagen[pixel *4 + 1] = 255;// canal G
		imagen[pixel *4 + 2] = 255;// canal B
		imagen[pixel *4 + 3] = 255;	
	}
}

__host__ void check_CUDA_Error(const char *mensaje){
	hipError_t error;
	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess){
		printf("ERROR %d: %s (%s)\n", error, hipGetErrorString(error),
		mensaje );
		printf("\npulsa INTRO para finalizar...");
		fflush(stdin);
		char tecla =getchar();
		exit(-1);
	}
}


// MAIN:
int main(int argc, char** argv)
{
	// declaracion del bitmap
	CPUBitmap bitmap( DIM, DIM );
	// tamaño en bytes
	size_t size = bitmap.image_size();
	// reserva en el host
	unsigned char *host_bitmap = bitmap.get_ptr();
	// reserva en el device
	unsigned char *dev_bitmap;
	hipMalloc( (void**)&dev_bitmap, size );
	// generamos el bitmap
	dim3 Nbloques(DIM/16,DIM/16);
	dim3 hilosB(16,16);
	kernel<<<Nbloques,hilosB>>>( dev_bitmap );
	// recogemos el bitmap desde la GPU para visualizarlo
	hipMemcpy( host_bitmap, dev_bitmap, size, hipMemcpyDeviceToHost );
	// liberacion de recursos
	hipFree( dev_bitmap );
	// visualizacion y salida
	bitmap.display_and_exit();
	printf("\n...Ejecucion Terminada...");
	
	return 0;
}

